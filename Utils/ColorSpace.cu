#include "hip/hip_runtime.h"
/*
* Copyright 2017-2021 NVIDIA Corporation.  All rights reserved.
*
* Please refer to the NVIDIA end user license agreement (EULA) associated
* with this source code for terms and conditions that govern your use of
* this software. Any use, reproduction, disclosure, or distribution of
* this software and related documentation outside the terms of the EULA
* is strictly prohibited.
*
*/

#include "ColorSpace.h"

__constant__ float matYuv2Rgb[3][3];
__constant__ float matRgb2Yuv[3][3];

bool setMatFlag=false;
void inline GetConstants(int iMatrix, float &wr, float &wb, int &black, int &white, int &max) {
    black = 16; white = 235;
    max = 255;

    switch (iMatrix)
    {
    case ColorSpaceStandard_BT709:
    default:
        wr = 0.2126f; wb = 0.0722f;
        break;

    case ColorSpaceStandard_FCC:
        wr = 0.30f; wb = 0.11f;
        break;

    case ColorSpaceStandard_BT470:
    case ColorSpaceStandard_BT601:
        wr = 0.2990f; wb = 0.1140f;
        break;

    case ColorSpaceStandard_SMPTE240M:
        wr = 0.212f; wb = 0.087f;
        break;

    case ColorSpaceStandard_BT2020:
    case ColorSpaceStandard_BT2020C:
        wr = 0.2627f; wb = 0.0593f;
        // 10-bit only
        black = 64 << 6; white = 940 << 6;
        max = (1 << 16) - 1;
        break;
    }
}

void __global__ add_kernel(float* a, float* b, float* c, int size) {
	const int idx = blockIdx.x * (blockDim.x * blockDim.y) + threadIdx.x + threadIdx.y * blockDim.x;
	//const int idx = threadIdx.x + threadIdx.y * blockDim.x;
	if (idx < size) {
		c[idx] = a[idx] + b[idx];
	}
}

void SetMatYuv2Rgb(int iMatrix) {
    if(setMatFlag){
        return;
    }
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        1.0f, 0.0f, (1.0f - wr) / 0.5f,
        1.0f, -wb * (1.0f - wb) / 0.5f / (1 - wb - wr), -wr * (1 - wr) / 0.5f / (1 - wb - wr),
        1.0f, (1.0f - wb) / 0.5f, 0.0f,
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * max / (white - black) * mat[i][j]);
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(matYuv2Rgb), mat, sizeof(mat));
    setMatFlag=true;
}

void SetMatRgb2Yuv(int iMatrix) {
    float wr, wb;
    int black, white, max;
    GetConstants(iMatrix, wr, wb, black, white, max);
    float mat[3][3] = {
        wr, 1.0f - wb - wr, wb,
        -0.5f * wr / (1.0f - wb), -0.5f * (1 - wb - wr) / (1.0f - wb), 0.5f,
        0.5f, -0.5f * (1.0f - wb - wr) / (1.0f - wr), -0.5f * wb / (1.0f - wr),
    };
    for (int i = 0; i < 3; i++) {
        for (int j = 0; j < 3; j++) {
            mat[i][j] = (float)(1.0 * (white - black) / max * mat[i][j]);
        }
    }
    hipMemcpyToSymbol(HIP_SYMBOL(matRgb2Yuv), mat, sizeof(mat));
}

template<class T>
__device__ static T Clamp(T x, T lower, T upper) {
    return x < lower ? lower : (x > upper ? upper : x);
}

template<class Rgb, class YuvUnit>
__device__ inline Rgb YuvToRgbForPixel(YuvUnit y, YuvUnit u, YuvUnit v) {
    const int 
        low = 1 << (sizeof(YuvUnit) * 8 - 4),
        mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    float fy = (int)y - low, fu = (int)u - mid, fv = (int)v - mid;
    const float maxf = (1 << sizeof(YuvUnit) * 8) - 1.0f;
    YuvUnit 
        r = (YuvUnit)Clamp(matYuv2Rgb[0][0] * fy + matYuv2Rgb[0][1] * fu + matYuv2Rgb[0][2] * fv, 0.0f, maxf),
        g = (YuvUnit)Clamp(matYuv2Rgb[1][0] * fy + matYuv2Rgb[1][1] * fu + matYuv2Rgb[1][2] * fv, 0.0f, maxf),
        b = (YuvUnit)Clamp(matYuv2Rgb[2][0] * fy + matYuv2Rgb[2][1] * fu + matYuv2Rgb[2][2] * fv, 0.0f, maxf);
    
    Rgb rgb{};
    const int nShift = abs((int)sizeof(YuvUnit) - (int)sizeof(rgb.c.r)) * 8;
    if (sizeof(YuvUnit) >= sizeof(rgb.c.r)) {
        rgb.c.r = r >> nShift;
        rgb.c.g = g >> nShift;
        rgb.c.b = b >> nShift;
    } else {
        rgb.c.r = r << nShift;
        rgb.c.g = g << nShift;
        rgb.c.b = b << nShift;
    }
    return rgb;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void YuvToRgbKernel(
    uint8_t *pYuv, 
    int nYuvPitch, 
    uint8_t *pRgb, 
    int nRgbPitch, 
    int nWidth, 
    int nHeight) 
{
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    *(RgbIntx2 *)pDst = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y).d,
    };
    *(RgbIntx2 *)(pDst + nRgbPitch) = RgbIntx2 {
        YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y).d, 
        YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y).d,
    };
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void Yuv444ToRgbKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgb, int nRgbPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    if (x + 1 >= nWidth || y  >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    uint8_t *pDst = pRgb + x * sizeof(Rgb) + y * nRgbPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (nHeight * nYuvPitch));
    YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * nHeight * nYuvPitch));

    *(RgbIntx2 *)pDst = RgbIntx2{
        YuvToRgbForPixel<Rgb>(l0.x, ch1.x, ch2.x).d,
        YuvToRgbForPixel<Rgb>(l0.y, ch1.y, ch2.y).d,
    };
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void YuvToRgbPlanarKernel(
    uint8_t *pYuv, 
    int nYuvPitch, 
    uint8_t *pRgbp, 
    int nRgbpPitch, 
    int nWidth, 
    int nHeight) 
{
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 l1 = *(YuvUnitx2 *)(pSrc + nYuvPitch);
    YuvUnitx2 ch = *(YuvUnitx2 *)(pSrc + (nHeight - y / 2) * nYuvPitch);

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch.x, ch.y),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch.x, ch.y),
        rgb2 = YuvToRgbForPixel<Rgb>(l1.x, ch.x, ch.y),
        rgb3 = YuvToRgbForPixel<Rgb>(l1.y, ch.x, ch.y);

    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.x, rgb1.v.x};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.x, rgb3.v.x};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.y, rgb1.v.y};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.y, rgb3.v.y};
    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2 {rgb0.v.z, rgb1.v.z};
    *(RgbUnitx2 *)(pDst + nRgbpPitch) = RgbUnitx2 {rgb2.v.z, rgb3.v.z};
}

template<class YuvUnitx2, class Rgb, class RgbUnitx2>
__global__ static void Yuv444ToRgbPlanarKernel(uint8_t *pYuv, int nYuvPitch, uint8_t *pRgbp, int nRgbpPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y);
    if (x + 1 >= nWidth || y >= nHeight) {
        return;
    }

    uint8_t *pSrc = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;

    YuvUnitx2 l0 = *(YuvUnitx2 *)pSrc;
    YuvUnitx2 ch1 = *(YuvUnitx2 *)(pSrc + (nHeight * nYuvPitch));
    YuvUnitx2 ch2 = *(YuvUnitx2 *)(pSrc + (2 * nHeight * nYuvPitch));

    Rgb rgb0 = YuvToRgbForPixel<Rgb>(l0.x, ch1.x, ch2.x),
        rgb1 = YuvToRgbForPixel<Rgb>(l0.y, ch1.y, ch2.y);


    uint8_t *pDst = pRgbp + x * sizeof(RgbUnitx2) / 2 + y * nRgbpPitch;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.x, rgb1.v.x };

    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.y, rgb1.v.y };

    pDst += nRgbpPitch * nHeight;
    *(RgbUnitx2 *)pDst = RgbUnitx2{ rgb0.v.z, rgb1.v.z };
}

template <class COLOR32>
void Nv12ToColor32(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    // std::cout<<"Sync YuvToRgbKernel stream "<<std::endl;
    SetMatYuv2Rgb(iMatrix);
    // std::cout<<"iMatrix "<<iMatrix<<std::endl;
    YuvToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void Nv12ToColor32Async(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix,hipStream_t &stream) {
    
    // std::cout<<"Async YuvToRgbKernel stream "<<std::endl;
    SetMatYuv2Rgb(iMatrix);
    // std::cout<<"iMatrix "<<iMatrix<<std::endl;
    // YuvToRgbKernel<uchar2, COLOR32, uint2>
    //     <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2),0,stream>>>
    //     (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
    YuvToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 63) / 32 / 2), dim3(32, 32),0,stream>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "YuvToRgbKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
}

template <class COLOR64>
void Nv12ToColor64(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<uchar2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444ToColor32(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<uchar2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void YUV444ToColor64(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<uchar2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void P016ToColor32(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<ushort2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void P016ToColor64(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbKernel<ushort2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444P16ToColor32(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<ushort2, COLOR32, uint2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR64>
void YUV444P16ToColor64(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbKernel<ushort2, COLOR64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgra, nBgraPitch, nWidth, nHeight);
}

template <class COLOR32>
void Nv12ToColorPlanar(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbPlanarKernel<uchar2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpNv12, nNv12Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void P016ToColorPlanar(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    YuvToRgbPlanarKernel<ushort2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpP016, nP016Pitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444ToColorPlanar(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbPlanarKernel<uchar2, COLOR32, uchar2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >>>
        (dpYUV444, nPitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

template <class COLOR32>
void YUV444P16ToColorPlanar(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix) {
    SetMatYuv2Rgb(iMatrix);
    Yuv444ToRgbPlanarKernel<ushort2, COLOR32, uchar2>
        << <dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2), dim3(32, 2) >> >
        (dpYUV444, nPitch, dpBgrp, nBgrpPitch, nWidth, nHeight);
}

// Explicit Instantiation
template void Nv12ToColor32<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor32<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor32Async<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix,hipStream_t &stream);
template void Nv12ToColor32Async<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix,hipStream_t &stream);
template void Nv12ToColor64<BGRA64>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColor64<RGBA64>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor32<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor32<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor64<BGRA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColor64<RGBA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor32<BGRA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor32<RGBA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor64<BGRA64>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColor64<RGBA64>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor32<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor32<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor64<BGRA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColor64<RGBA64>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgra, int nBgraPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColorPlanar<BGRA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void Nv12ToColorPlanar<RGBA32>(uint8_t *dpNv12, int nNv12Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColorPlanar<BGRA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void P016ToColorPlanar<RGBA32>(uint8_t *dpP016, int nP016Pitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColorPlanar<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444ToColorPlanar<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColorPlanar<BGRA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);
template void YUV444P16ToColorPlanar<RGBA32>(uint8_t *dpYUV444, int nPitch, uint8_t *dpBgrp, int nBgrpPitch, int nWidth, int nHeight, int iMatrix);

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToY(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit low = 1 << (sizeof(YuvUnit) * 8 - 4);
    return matRgb2Yuv[0][0] * r + matRgb2Yuv[0][1] * g + matRgb2Yuv[0][2] * b + low;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToU(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[1][0] * r + matRgb2Yuv[1][1] * g + matRgb2Yuv[1][2] * b + mid;
}

template<class YuvUnit, class RgbUnit>
__device__ inline YuvUnit RgbToV(RgbUnit r, RgbUnit g, RgbUnit b) {
    const YuvUnit mid = 1 << (sizeof(YuvUnit) * 8 - 1);
    return matRgb2Yuv[2][0] * r + matRgb2Yuv[2][1] * g + matRgb2Yuv[2][2] * b + mid;
}

template<class YuvUnitx2, class Rgb, class RgbIntx2>
__global__ static void RgbToYuvKernel(uint8_t *pRgb, int nRgbPitch, uint8_t *pYuv, int nYuvPitch, int nWidth, int nHeight) {
    int x = (threadIdx.x + blockIdx.x * blockDim.x) * 2;
    int y = (threadIdx.y + blockIdx.y * blockDim.y) * 2;
    if (x + 1 >= nWidth || y + 1 >= nHeight) {
        return;
    }

    uint8_t *pSrc = pRgb + x * sizeof(Rgb) + y * nRgbPitch;
    RgbIntx2 int2a = *(RgbIntx2 *)pSrc;
    RgbIntx2 int2b = *(RgbIntx2 *)(pSrc + nRgbPitch);

    Rgb rgb[4] = {int2a.x, int2a.y, int2b.x, int2b.y};
    decltype(Rgb::c.r)
        r = (rgb[0].c.r + rgb[1].c.r + rgb[2].c.r + rgb[3].c.r) / 4,
        g = (rgb[0].c.g + rgb[1].c.g + rgb[2].c.g + rgb[3].c.g) / 4,
        b = (rgb[0].c.b + rgb[1].c.b + rgb[2].c.b + rgb[3].c.b) / 4;

    uint8_t *pDst = pYuv + x * sizeof(YuvUnitx2) / 2 + y * nYuvPitch;
    *(YuvUnitx2 *)pDst = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[0].c.r, rgb[0].c.g, rgb[0].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[1].c.r, rgb[1].c.g, rgb[1].c.b),
    };
    *(YuvUnitx2 *)(pDst + nYuvPitch) = YuvUnitx2 {
        RgbToY<decltype(YuvUnitx2::x)>(rgb[2].c.r, rgb[2].c.g, rgb[2].c.b),
        RgbToY<decltype(YuvUnitx2::x)>(rgb[3].c.r, rgb[3].c.g, rgb[3].c.b),
    };
    *(YuvUnitx2 *)(pDst + (nHeight - y / 2) * nYuvPitch) = YuvUnitx2 {
        RgbToU<decltype(YuvUnitx2::x)>(r, g, b), 
        RgbToV<decltype(YuvUnitx2::x)>(r, g, b),
    };
}

void Bgra64ToP016(uint8_t *dpBgra, int nBgraPitch, uint8_t *dpP016, int nP016Pitch, int nWidth, int nHeight, int iMatrix) {
    SetMatRgb2Yuv(iMatrix);
    RgbToYuvKernel<ushort2, BGRA64, ulonglong2>
        <<<dim3((nWidth + 63) / 32 / 2, (nHeight + 3) / 2 / 2), dim3(32, 2)>>>
        (dpBgra, nBgraPitch, dpP016, nP016Pitch, nWidth, nHeight);
}

__global__ void RGBA2RGB_kernel(uint8_t *pRGBA,uint8_t *pRGB,int rows,int cols){
    const int id=blockIdx.x*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x;
    if(id<rows*cols){
        pRGB[id*3]=pRGBA[id*4];
        pRGB[id*3+1]=pRGBA[id*4+1];
        pRGB[id*3+2]=pRGBA[id*4+2];
    }
}

__global__ void RGB2RGBA_kernel(uint8_t *pRGB,uint8_t *pRGBA,int rows,int cols){
    const int id=blockIdx.x*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x;
    if(id<rows*cols){
        pRGBA[id*4]=pRGB[id*3];
        pRGBA[id*4+1]=pRGB[id*3+1];
        pRGBA[id*4+2]=pRGB[id*3+2];
        pRGBA[id*4+3]=0;
    }
}

__global__ void bgr2yuv420(
    uint8_t *Bgra,
    uint8_t *Yuv,
    // uint8_t *u,
    int rows,int cols)
{
    const int idx=blockIdx.x*blockDim.x*blockDim.y+threadIdx.y*blockDim.x+threadIdx.x;
    int dx=idx%cols;
    int dy=idx/cols;
    // int stride=gridDim.x*blockDim.x;
    // int idx=threadIdx.x+blockIdx.x*blockDim.x;


    short3 yuv16;
    char3 yuv8;
    int ix;
    int iy;
    if(idx<rows*cols){
        yuv16.x=66*Bgra[idx*4+2]+ 129*Bgra[idx*4+1]+25*Bgra[idx*4];
        yuv16.y=-38*Bgra[idx*4+2] -74*Bgra[idx*4+1]+112*Bgra[idx*4];
        yuv16.z=112*Bgra[idx*4+2] -94*Bgra[idx*4+1] -18*Bgra[idx*4];

        yuv8.x=(yuv16.x>>8)+16;
        yuv8.y=(yuv16.y>>8)+128;
        yuv8.z=(yuv16.z>>8)+128;

        Yuv[idx]=yuv8.x;

        
        // ix=dx/2;
        iy=dy/4;
        ix=dx/2+(dy&1)*cols/2;
        // if(dy%2==0){
        //     ix=dx/2;
        // }else{
        //     // ix=dx/2+cols/2;
        //     ix=dx/2+(dy&1)*cols/2;
        // }
        // u[iy*cols+ix]=yuv8.y;
        // Yuv[cols*rows+dy*cols/4+dx/2+(dy&1)*cols/2]=yuv8.y;
        Yuv[cols*rows+iy*cols+ix]=yuv8.y;
        Yuv[cols*rows*5/4+iy*cols+ix]=yuv8.z;
        // if(dx%2==0 && dy%2==0){
        //     // Yuv[rows*cols+cols*dy/4+dx]=yuv8.y;
        //     // Yuv[rows*cols*5/4+cols*dy/4+dx]=yuv8.z;
        //     Yuv[rows*cols+idx/4]=yuv8.y;
        //     // Yuv[rows*cols*5/4+idx/4]=yuv8.z;
        // }
        // else if(dx%2==0){
        //     // Yuv[rows*cols+cols*dy/4+dx]=yuv8.y;
        //     // Yuv[rows*cols*5/4+cols*dy/4+dx]=yuv8.z;
        //     // Yuv[rows*cols+idx/4]=yuv8.y;
        //     Yuv[rows*cols*5/4+idx/4]=yuv8.z;
        // }
        // Yuv[rows*cols+idx/2]=yuv8.y;
        // Yuv[rows*cols*5/4+idx/4]=yuv8.z;
        // Yuv[rows*cols*5/4+dy*cols/4]=yuv8.z;

        // Yuv[rows*cols+cols*dy/4+dx]=yuv8.y/4;
        // Yuv[rows*cols*5/4+cols*dy/4+dx]=yuv8.z/4;
        // Yuv[rows*cols+cols*dy/2+dx]=yuv8.y;
        // Yuv[rows*cols+cols*dy/2+dx+1]=yuv8.z;
    }
}

void bgra2bgr(uint8_t *rgba,uint8_t *rgb,int rows,int cols){
    const int thread = 32;
	const int grid = (rows*cols + thread - 1) / (thread*thread);
	const dim3 blockSize(thread, thread);
	const dim3 gridSize(grid);
	RGBA2RGB_kernel<<<gridSize,blockSize>>>(rgba,rgb,rows,cols);
}
void bgr2bgra(uint8_t *rgb,uint8_t *rgba,int rows,int cols){
    const int thread = 32;
	const int grid = (rows*cols + thread - 1) / (thread*thread);
	const dim3 blockSize(thread, thread);
	const dim3 gridSize(grid);
	RGB2RGBA_kernel<<<gridSize,blockSize>>>(rgb,rgba,rows,cols);
}

void bgra2bgrAsync(uint8_t *rgba,uint8_t *rgb,int rows,int cols,hipStream_t &stream){
    const int thread = 32;
	const int grid = (rows*cols + thread - 1) / (thread*thread);
	const dim3 blockSize(thread, thread);
	const dim3 gridSize(grid);
	RGBA2RGB_kernel<<<gridSize,blockSize,0,stream>>>(rgba,rgb,rows,cols);
}

void bgr2bgraAsync(uint8_t *rgb,uint8_t *rgba,int rows,int cols,hipStream_t &stream){
    const int thread = 32;
	const int grid = (rows*cols + thread - 1) / (thread*thread);
	const dim3 blockSize(thread, thread);
	const dim3 gridSize(grid);
	RGB2RGBA_kernel<<<gridSize,blockSize,0,stream>>>(rgb,rgba,rows,cols);
}

void Bgra2Yuv(uint8_t *bgra,uint8_t *yuv,int rows,int cols){
    const int thread = 32;
	const int grid = (rows*cols + thread - 1) / (thread*thread);
	const dim3 blockSize(thread, thread);
	const dim3 gridSize(grid);

    // int *a;
    
    // a=(int*)malloc(sizeof(int)*2);
    // memset(a,0,sizeof(int)*2);
    bgr2yuv420<<<gridSize,blockSize>>>(bgra,yuv,rows,cols);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "convertPixelFormat launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

    // std::cout<<"(a-> "<<a[0]<<" ,b-> "<<a[1]<<std::endl;
}
void Bgra2YuvAsync(uint8_t *bgra,uint8_t *yuv,int rows,int cols,hipStream_t &stream){
    const int thread = 32;
	const int grid = (rows*cols + thread - 1) / (thread*thread);
	const dim3 blockSize(thread, thread);
	const dim3 gridSize(grid);

    bgr2yuv420<<<gridSize,blockSize,0,stream>>>(bgra,yuv,rows,cols);
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "convertPixelFormat launch failed: %s\n", hipGetErrorString(cudaStatus));
    }

}
// void Bgra2Yuv1(uint8_t *bgra,uint8_t *yuv,uint8_t *u,int rows,int cols){
//     const int thread = 32;
// 	const int grid = (rows*cols + thread - 1) / (thread*thread);
// 	const dim3 blockSize(thread, thread);
// 	const dim3 gridSize(grid);

//     int *a;
    
//     a=(int*)malloc(sizeof(int)*2);
//     memset(a,0,sizeof(int)*2);
//     bgr2yuv420<<<gridSize,blockSize>>>(bgra,yuv,u,rows,cols,a);
//     hipError_t cudaStatus = hipGetLastError();
//     if (cudaStatus != hipSuccess)
//     {
//         fprintf(stderr, "convertPixelFormat launch failed: %s\n", hipGetErrorString(cudaStatus));
//     }

//     // std::cout<<"(a-> "<<a[0]<<" ,b-> "<<a[1]<<std::endl;
// }